/**
 * Copyright 2021-2023, XGBoost Contributors
 */
#include <cstdint>  // for int64_t

#include "../common/common.h"
#include "../common/device_helpers.cuh"  // for DefaultStream, CUDAEvent
#include "array_interface.h"
#include "xgboost/logging.h"

namespace xgboost {
void ArrayInterfaceHandler::SyncCudaStream(std::int64_t stream) {
  switch (stream) {
    case 0:
      /**
       * disallowed by the `__cuda_array_interface__`.  Quote:
       *
       *   This is disallowed as it would be ambiguous between None and the default
       *   stream, and also between the legacy and per-thread default streams. Any use
       *   case where 0 might be given should either use None, 1, or 2 instead for
       *   clarity.
       */
#ifndef XGBOOST_USE_HIP
      LOG(FATAL) << "Invalid stream ID in array interface: " << stream;
#endif
    case 1:
      // default legacy stream
      break;
    case 2:
      // default per-thread stream
    default: {
      dh::CUDAEvent e;
      e.Record(dh::CUDAStreamView{reinterpret_cast<hipStream_t>(stream)});
      dh::DefaultStream().Wait(e);
    }
  }
}

bool ArrayInterfaceHandler::IsCudaPtr(void const* ptr) {
  if (!ptr) {
    return false;
  }

#if defined(XGBOOST_USE_CUDA)
  hipPointerAttribute_t attr;
  auto err = hipPointerGetAttributes(&attr, ptr);
  // reset error
  CHECK_EQ(err, hipGetLastError());
  if (err == hipErrorInvalidValue) {
    // CUDA < 11
    return false;
  } else if (err == hipSuccess) {
    // CUDA >= 11
    switch (attr.type) {
      case cudaMemoryTypeUnregistered:
      case hipMemoryTypeHost:
        return false;
      default:
        return true;
    }
    return true;
  } else {
    // other errors, `hipErrorNoDevice`, `hipErrorInsufficientDriver` etc.
    return false;
  }
#elif defined(XGBOOST_USE_HIP)
  hipPointerAttribute_t attr;
  auto err = hipPointerGetAttributes(&attr, ptr);
  // reset error
  CHECK_EQ(err, hipGetLastError());
  if (err == hipErrorInvalidValue) {
    return false;
  } else if (err == hipSuccess) {
#if HIP_VERSION_MAJOR < 6
    switch (attr.memoryType) {
      case hipMemoryTypeHost:
        return false;
      default:
        return true;
    }
#else
    switch (attr.type) {
      case hipMemoryTypeUnregistered:
      case hipMemoryTypeHost:
        return false;
      default:
        return true;
    }
#endif
    return true;
  } else {
    return false;
  }
#endif
}
}  // namespace xgboost
