/**
 * Copyright 2018-2024, XGBoost contributors
 */
#if defined(XGBOOST_USE_CUDA)
#include <thrust/system/cuda/error.h>
#elif defined(XGBOOST_USE_HIP)
#include <thrust/system/hip/error.h>
#endif
#include <thrust/system_error.h>

#include "common.h"
#include "cuda_to_hip.h"

namespace dh {
void ThrowOnCudaError(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    std::string f;
    if (file != nullptr) {
      f = file;
    }
    LOG(FATAL) << thrust::system_error(code, thrust::cuda_category(),
                                       f + ": " + std::to_string(line))
                      .what();
  }
}
}  // namespace dh
